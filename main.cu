﻿#include "CosmosCuda.cuh"
// Change these values to utilize your gpu better (currently they are tuned for RTX4070)
// Constants::BLOCKS
// Constants::THREADS
// Change this to another power-of-2 to tune accuracy of long-ranged forces. Short-ranged forces are not calculated currently (todo).
// Constants::N
int main() {
    const int numNbodySimulationsPerRender = 2;
    // 20 Bytes per particle is allocated.
    const int numParticles = 1000 * 1000 * 10;
    // cuda device index
    const int device = 0;
    // true: more performance
    const bool lowAccuracy = false;
    // Window width/height
    const int w = 1200;
    const int h = 1200;
    Universe cosmos(numParticles, device, lowAccuracy, w, h);
    
    while (true) {
        cosmos.startBenchmark();
        for (int i = 0; i < numNbodySimulationsPerRender; i++) {
            cosmos.nBody();
        }
        cosmos.stopBenchmark();
        cosmos.sync(numNbodySimulationsPerRender);
        cosmos.render();
        if (cv::waitKey(1) == 27) {
            break;
        }
    }
    return 0;
}